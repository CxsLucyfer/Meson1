#include "hip/hip_runtime.h"
#include <stdio.h>

#include "b.h"

__global__ void foo(void)
{
    __shared__ int a[N];
    a[threadIdx.x] = threadIdx.x;

    __syncthreads();

    g[threadIdx.x] = a[blockDim.x - threadIdx.x - 1];

    bar();
}

int main(void)
{
    unsigned int i;
    int *dg, hg[N];
    int sum = 0;

    foo<<<1, N>>>();

    if (hipGetSymbolAddress((void**)&dg, g)) {
        printf("couldn't get the symbol addr\n");
        return 1;
    }
    if (hipMemcpy(hg, dg, N * sizeof(int), hipMemcpyDeviceToHost)) {
        printf("couldn't memcpy\n");
        return 1;
    }

    for (i = 0; i < N; i++) {
        sum += hg[i];
    }
    if (sum == 36) {
        printf("PASSED\n");
    } else {
        printf("FAILED (%d)\n", sum);
    }

    return 0;
}
